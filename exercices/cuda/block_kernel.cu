
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void init_timing(struct timeval* tstart)
{
    gettimeofday(tstart, NULL);
}

float ellapsed_time(struct timeval tstart)
{
    struct timeval tmp;
    long long diff;
    gettimeofday(&tmp, NULL);
    diff = tmp.tv_usec - tstart.tv_usec;
    diff += (tmp.tv_sec - tstart.tv_sec) * 1000000;
    return ((float)diff*1.0e-6);
}

__global__ void loop_workshare_kernel(float *A, float *B, int size, int rad)
{
	//int i = threadIdx.x;
	int i = blockIdx.z * blockDim.z + threadIdx.z;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	
	int dx, dy, dz;
	
	float temp_val = 0.0;
	
	if(i >= rad && i < size - rad)
	{
		if(j >= rad && j < size - rad)
		{
			if(k >= rad && k < size - rad)
			{	
				for(dx = -rad; dx <= rad; dx++)
				{
					for(dy = -rad; dy <= rad; dy++)
					{
						for(dz = -rad; dz <= rad; dz++)
						{
								temp_val += A[(i+dx)*size*size + (j+dy)*size + (k+dz)];
						}
					}
				}
				
				B[i*size*size + j*size + k] = temp_val;
			}
		}
	}
	
}


int main()
{
	int i, j, k;
	int N = 512, rad = 2;
	float *A, *B, *device_A, *device_B;
	
	struct timeval timer;
	
	A = (float*) malloc(N*N*N*sizeof(float));
	B = (float*) malloc(N*N*N*sizeof(float));
	
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			for(k = 0; k < N; k++)
			{
				A[i*N*N + j*N + k] = i + j + k;
				B[i*N*N + j*N + k] = 0.0;
			}
		}
	}
	
	hipMalloc(&device_A, N*N*N*sizeof(float));
	hipMalloc(&device_B, N*N*N*sizeof(float));
	hipMemcpy(device_A, A, N*N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(device_B, B, N*N*N*sizeof(float), hipMemcpyHostToDevice);
	
	//WARNNG : dim_a * dim_b * dim_c < 1024
	// AND dim_a < 1024, dimb_b < 1024 & dim_c < 64
	//Must try to mixmise gpu occupancy
	dim3 threadsPerBlock(32,4,2);
	dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
					(N + threadsPerBlock.y - 1) / threadsPerBlock.y,
		 			(N + threadsPerBlock.z - 1) / threadsPerBlock.z);
	
	init_timing(&timer);
	
	loop_workshare_kernel<<< numBlocks, threadsPerBlock >>>(device_A, device_B, N, rad);
	hipDeviceSynchronize();
	
	printf("Kernel time : %f\n", ellapsed_time(timer));
	
	hipMemcpy(B, device_B, N*N*N*sizeof(float), hipMemcpyDeviceToHost);
	
	printf("%f\n", B[N*N*N/2 + N*N/2 + N/2]);
	
	
	free(A);
	free(B);
	hipFree(device_A);
	hipFree(device_B);
	
	exit(EXIT_SUCCESS);
}
