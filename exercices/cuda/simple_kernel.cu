
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

void init_timing(struct timeval* tstart)
{
    gettimeofday(tstart, NULL);
}

float ellapsed_time(struct timeval tstart)
{
    struct timeval tmp;
    long long diff;
    gettimeofday(&tmp, NULL);
    diff = tmp.tv_usec - tstart.tv_usec;
    diff += (tmp.tv_sec - tstart.tv_sec) * 1000000;
    return ((float)diff*1.0e-6);
}

int cu_threads = 128;

__global__ void op_kernel(float *tab, int size)
{
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float prev, next;
	
	if(i < size)
	{
		if(i == 0)
			prev = 0;
		else
			prev = tab[i-1];
			
		if(i == size-1)
			next = 0;
		else
			next = tab[i+1];
		
		tab[i] = tab[i] * prev / next;
	}
	
}


int main()
{
	int i;
	int N = 65536;
	float *table, *device_table;
	float prev, next;
	
	struct timeval timer;
	
	int cu_blocks;
	
	table = (float*) malloc(N*sizeof(float));
	
	for(i = 0; i < N; i++)
	{
		table[i] = i;
	}
	
	hipMalloc(&device_table, N*sizeof(float));
	hipMemcpy(device_table, table, N*sizeof(float), hipMemcpyHostToDevice);
	
	cu_blocks = (N + cu_threads - 1) / cu_threads;
	
	init_timing(&timer);
	
	op_kernel<<< cu_blocks, cu_threads >>>(device_table, N);
	hipDeviceSynchronize();
	
	printf("Kernel time : %f\n", ellapsed_time(timer));
	
	hipMemcpy(table, device_table, N*sizeof(float), hipMemcpyDeviceToHost);
	
	init_timing(&timer);
	
	for(i = 0; i < N; i++)
	{
		if(i == 0)
			prev = 0;
		else
			prev = table[i-1];
			
		if(i == N-1)
			next = 0;
		else
			next = table[i+1];
		
		table[i] = table[i] * prev / next;
	}
	
	printf("CPU time : %f\n", ellapsed_time(timer));
	for(i = 0; i < N; i++)
		printf("%d\n",table[i]);
	
	free(table);
	hipFree(device_table);
	
	exit(EXIT_SUCCESS);
}
