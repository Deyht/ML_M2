
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <cblas.h>
#include <hipblas.h>

//##########################################################
// In code timer function
//##########################################################


void init_timing(struct timeval* tstart)
{
    gettimeofday(tstart, NULL);
}

float ellapsed_time(struct timeval tstart)
{
    struct timeval tmp;
    long long diff;
    gettimeofday(&tmp, NULL);
    diff = tmp.tv_usec - tstart.tv_usec;
    diff += (tmp.tv_sec - tstart.tv_sec) * 1000000;
    return ((float)diff*1.0e-6);
}


//##########################################################

extern "C"
{
void blas_fct(void);
}

void blas_fct(void)
{
	int i;
	int N = 8192;
	float *A, *B, *C;
	float *d_A, *d_B, *d_C;
	float alpha = 1.0, beta=0.0;
	
	struct timeval timer;
	hipblasHandle_t cu_handle;
	
	if(hipblasCreate(&cu_handle) != HIPBLAS_STATUS_SUCCESS) 
	{
		printf("GPU handle create fail\n");
		exit(EXIT_FAILURE);
	}
	
	
	A = (float*) malloc(N*N*sizeof(float));
	B = (float*) malloc(N*N*sizeof(float));
	C = (float*) malloc(N*N*sizeof(float));
	
	for(i = 0; i < N*N; i++)
	{
		A[i] = (i%50) * 0.1;
		B[i] = (i%25) * 1.3;
		C[i] = 0.0;
	}
	
	hipMalloc(&d_A, N*N*sizeof(float));
	hipMalloc(&d_B, N*N*sizeof(float));
	hipMalloc(&d_C, N*N*sizeof(float));
	
	hipMemcpy(d_A, A, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, N*N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, N*N*sizeof(float), hipMemcpyHostToDevice);
	
	
	init_timing(&timer);
	
	cblas_sgemm(CblasColMajor, CblasNoTrans, CblasNoTrans, N, N, N, alpha, A, N, B, N, beta, C, N);
	
	printf("CPU time : %f\n", ellapsed_time(timer));
	
	printf("%f\n", C[N*N/2 + N/2]);

	for(i = 0; i < N*N; i++)
	{
		C[i] = 0.0;
	}

	init_timing(&timer);
	
	hipblasSgemm(cu_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, N);
	hipDeviceSynchronize();
	printf("CUDA time : %f\n", ellapsed_time(timer));

	hipMemcpy(C, d_C, N*N*sizeof(float), hipMemcpyDeviceToHost);
	printf("%f\n", C[N*N/2 + N/2]);
	
	free(A);
	free(B);
	free(C);
}


