
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void add_kernel(int *tab, int size)
{
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < size)
	{
		tab[i]*=2;
	}
	
}


int main()
{
	int i;
	int N = 64;
	int *table, *device_table;
	
	table = (int*) malloc(N*sizeof(int));
	
	for(i = 0; i < N; i++)
	{
		table[i] = i;
	}
	
	hipMalloc(&device_table, N*sizeof(int));
	hipMemcpy(device_table, table, N*sizeof(int), hipMemcpyHostToDevice);
	
	add_kernel<<< 1, N>>>(device_table, N);
	
	hipMemcpy(table, device_table, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for(i = 0; i < N; i++)
	{
		printf("%d\n",table[i]);
	}
	
	
	free(table);
	hipFree(device_table);
	
	exit(EXIT_SUCCESS);
}
