
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void hello_kernel(void)
{
	//int i = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int b = blockIdx.x;
	
	printf("Hello from block : %d, threadId : %d\n", b, i);
}


int main()
{
	
	hello_kernel<<< 4, 16>>>();
	
	//printf from device are not automatically flushed
	hipDeviceSynchronize();
	
	exit(EXIT_SUCCESS);
}
